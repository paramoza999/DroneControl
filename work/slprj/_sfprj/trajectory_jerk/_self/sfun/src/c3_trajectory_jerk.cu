/* Include files */

#include "trajectory_jerk_sfun.h"
#include "c3_trajectory_jerk.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include "stdio.h"
#include "string.h"
#include "stdlib.h"

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void c3_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance, real_T c3_b_xdd_des, real_T c3_b_ydd_des, real_T c3_b_psi,
  real_T *c3_b_phi_des, real_T *c3_b_theta_des);
static void initialize_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance);
static void initialize_params_c3_trajectory_jerk
  (SFc3_trajectory_jerkInstanceStruct *chartInstance);
static void mdl_start_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance);
static void mdl_terminate_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct *
  chartInstance);
static void mdl_setup_runtime_resources_c3_trajectory_jerk
  (SFc3_trajectory_jerkInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c3_trajectory_jerk
  (SFc3_trajectory_jerkInstanceStruct *chartInstance);
static void enable_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance);
static void disable_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance);
static void sf_gateway_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance);
static void ext_mode_exec_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct *
  chartInstance);
static const mxArray *get_sim_state_c3_trajectory_jerk
  (SFc3_trajectory_jerkInstanceStruct *chartInstance);
static void set_sim_state_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct *
  chartInstance, const mxArray *c3_st);
static real_T c3_emlrt_marshallIn(SFc3_trajectory_jerkInstanceStruct
  *chartInstance, const mxArray *c3_nullptr, const char_T *c3_identifier);
static real_T c3_b_emlrt_marshallIn(SFc3_trajectory_jerkInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId);
static void c3_raiseCudaError(SFc3_trajectory_jerkInstanceStruct *chartInstance,
  int32_T c3_errCode, const char_T *c3_file, uint32_T c3_line, const char_T
  *c3_errorName, const char_T *c3_errorString);
static void c3_checkCudaError(SFc3_trajectory_jerkInstanceStruct *chartInstance,
  hipError_t c3_errCode, const char_T *c3_file, uint32_T c3_line);
static void init_dsm_address_info(SFc3_trajectory_jerkInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc3_trajectory_jerkInstanceStruct
  *chartInstance);

/* Function Definitions */
static void c3_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance, real_T c3_b_xdd_des, real_T c3_b_ydd_des, real_T c3_b_psi,
  real_T *c3_b_phi_des, real_T *c3_b_theta_des)
{
  const mxArray *c3_b_y = NULL;
  const mxArray *c3_y = NULL;
  *c3_b_phi_des = -0.1019367991845056 * (c3_b_xdd_des * muDoubleScalarSin
    (c3_b_psi) - c3_b_ydd_des * muDoubleScalarCos(c3_b_psi));
  sf_mex_printf("%s =\\n", "phi_des");
  c3_y = NULL;
  sf_mex_assign(&c3_y, sf_mex_create("y", c3_b_phi_des, 0, 0U, 0, 0U, 0), false);
  sf_mex_call(chartInstance->c3_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c3_y);
  *c3_b_theta_des = -0.1019367991845056 * (c3_b_xdd_des * muDoubleScalarCos
    (c3_b_psi) + c3_b_ydd_des * muDoubleScalarSin(c3_b_psi));
  sf_mex_printf("%s =\\n", "theta_des");
  c3_b_y = NULL;
  sf_mex_assign(&c3_b_y, sf_mex_create("y", c3_b_theta_des, 0, 0U, 0, 0U, 0),
                false);
  sf_mex_call(chartInstance->c3_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c3_b_y);
}

static void initialize_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance)
{
  emlrtInitGPU(chartInstance->c3_fEmlrtCtx);
  hipGetLastError();
  emlrtLicenseCheckR2022a(chartInstance->c3_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void initialize_params_c3_trajectory_jerk
  (SFc3_trajectory_jerkInstanceStruct *chartInstance)
{
}

static void mdl_start_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct *
  chartInstance)
{
  hipError_t c3_errCode;
  c3_errCode = hipGetLastError();
  if (c3_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c3_errCode), (char_T *)
                       hipGetErrorName(c3_errCode), (char_T *)
                       hipGetErrorString(c3_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c3_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c3_trajectory_jerk
  (SFc3_trajectory_jerkInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, &chartInstance->c3_JITStateAnimation[0],
                        &chartInstance->c3_JITTransitionAnimation[0]);
}

static void mdl_cleanup_runtime_resources_c3_trajectory_jerk
  (SFc3_trajectory_jerkInstanceStruct *chartInstance)
{
}

static void enable_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct
  *chartInstance)
{
  real_T c3_d;
  real_T c3_d1;
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c3_JITTransitionAnimation[0] = 0U;
  c3_c3_trajectory_jerk(chartInstance, *chartInstance->c3_xdd_des,
                        *chartInstance->c3_ydd_des, *chartInstance->c3_psi,
                        &c3_d, &c3_d1);
  *chartInstance->c3_theta_des = c3_d1;
  *chartInstance->c3_phi_des = c3_d;
}

static void ext_mode_exec_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct *
  chartInstance)
{
}

static const mxArray *get_sim_state_c3_trajectory_jerk
  (SFc3_trajectory_jerkInstanceStruct *chartInstance)
{
  const mxArray *c3_b_y = NULL;
  const mxArray *c3_c_y = NULL;
  const mxArray *c3_st;
  const mxArray *c3_y = NULL;
  c3_st = NULL;
  c3_st = NULL;
  c3_y = NULL;
  sf_mex_assign(&c3_y, sf_mex_createcellmatrix(2, 1), false);
  c3_b_y = NULL;
  sf_mex_assign(&c3_b_y, sf_mex_create("y", chartInstance->c3_phi_des, 0, 0U, 0,
    0U, 0), false);
  sf_mex_setcell(c3_y, 0, c3_b_y);
  c3_c_y = NULL;
  sf_mex_assign(&c3_c_y, sf_mex_create("y", chartInstance->c3_theta_des, 0, 0U,
    0, 0U, 0), false);
  sf_mex_setcell(c3_y, 1, c3_c_y);
  sf_mex_assign(&c3_st, c3_y, false);
  return c3_st;
}

static void set_sim_state_c3_trajectory_jerk(SFc3_trajectory_jerkInstanceStruct *
  chartInstance, const mxArray *c3_st)
{
  const mxArray *c3_u;
  c3_u = sf_mex_dup(c3_st);
  *chartInstance->c3_phi_des = c3_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c3_u, 0)), "phi_des");
  *chartInstance->c3_theta_des = c3_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c3_u, 1)), "theta_des");
  sf_mex_destroy(&c3_u);
  sf_mex_destroy(&c3_st);
}

const mxArray *sf_c3_trajectory_jerk_get_eml_resolved_functions_info()
{
  const mxArray *c3_nameCaptureInfo = NULL;
  c3_nameCaptureInfo = NULL;
  sf_mex_assign(&c3_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1, 0U, 2, 0, 1), false);
  return c3_nameCaptureInfo;
}

static real_T c3_emlrt_marshallIn(SFc3_trajectory_jerkInstanceStruct
  *chartInstance, const mxArray *c3_nullptr, const char_T *c3_identifier)
{
  emlrtMsgIdentifier c3_thisId;
  real_T c3_y;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_y = c3_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_nullptr), &c3_thisId);
  sf_mex_destroy(&c3_nullptr);
  return c3_y;
}

static real_T c3_b_emlrt_marshallIn(SFc3_trajectory_jerkInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId)
{
  real_T c3_d;
  real_T c3_y;
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), &c3_d, 1, 0, 0U, 0, 0U, 0);
  c3_y = c3_d;
  sf_mex_destroy(&c3_u);
  return c3_y;
}

static void c3_raiseCudaError(SFc3_trajectory_jerkInstanceStruct *chartInstance,
  int32_T c3_errCode, const char_T *c3_file, uint32_T c3_line, const char_T
  *c3_errorName, const char_T *c3_errorString)
{
  emlrtRTEInfo c3_rtInfo;
  uint64_T c3_len;
  char_T *c3_brk;
  char_T *c3_fn;
  char_T *c3_pn;
  c3_len = strlen(c3_file);
  c3_pn = static_cast<char_T *>(calloc(c3_len + 1ULL, 1ULL));
  c3_fn = static_cast<char_T *>(calloc(c3_len + 1ULL, 1ULL));
  memcpy(c3_pn, c3_file, c3_len);
  memcpy(c3_fn, c3_file, c3_len);
  c3_brk = strrchr(c3_fn, '.');
  *c3_brk = '\x00';
  c3_brk = NULL;
  c3_brk = strrchr(c3_fn, '/');
  if (c3_brk == NULL) {
    c3_brk = strrchr(c3_fn, '\\');
  }

  if (c3_brk == NULL) {
    c3_brk = c3_fn;
  } else {
    c3_brk++;
  }

  c3_rtInfo.lineNo = c3_line;
  c3_rtInfo.colNo = 0;
  c3_rtInfo.fName = c3_brk;
  c3_rtInfo.pName = c3_pn;
  emlrtCUDAError(static_cast<uint32_T>(c3_errCode), (char_T *)c3_errorName,
                 (char_T *)c3_errorString, &c3_rtInfo,
                 chartInstance->c3_fEmlrtCtx);
}

static void c3_checkCudaError(SFc3_trajectory_jerkInstanceStruct *chartInstance,
  hipError_t c3_errCode, const char_T *c3_file, uint32_T c3_line)
{
  if (c3_errCode != hipSuccess) {
    c3_raiseCudaError(chartInstance, c3_errCode, c3_file, c3_line,
                      hipGetErrorName(c3_errCode), hipGetErrorString
                      (c3_errCode));
  }
}

static void init_dsm_address_info(SFc3_trajectory_jerkInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc3_trajectory_jerkInstanceStruct
  *chartInstance)
{
  chartInstance->c3_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c3_xdd_des = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c3_phi_des = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c3_ydd_des = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c3_psi = (real_T *)ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c3_theta_des = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c3_trajectory_jerk_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3245189126U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3573125509U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1290004641U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2872569169U);
}

mxArray *sf_c3_trajectory_jerk_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c3_trajectory_jerk_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c3_trajectory_jerk_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c3_trajectory_jerk(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wMjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig7o+lmw6GdH0i8A5RdkZManpBaD3Q8Lp4Hzhw"
    "LYHw4E/MGJ5g8QvyQjtSQR7BOEP0AAAIDPFtk="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c3_trajectory_jerk_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sMtLs8ofMki7VphfTfIASx";
}

static void sf_opaque_initialize_c3_trajectory_jerk(void *chartInstanceVar)
{
  initialize_params_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
  initialize_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c3_trajectory_jerk(void *chartInstanceVar)
{
  enable_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c3_trajectory_jerk(void *chartInstanceVar)
{
  disable_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c3_trajectory_jerk(void *chartInstanceVar)
{
  sf_gateway_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c3_trajectory_jerk(SimStruct* S)
{
  return get_sim_state_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c3_trajectory_jerk(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c3_trajectory_jerk(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc3_trajectory_jerkInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_trajectory_jerk_optimization_info();
    }

    mdl_cleanup_runtime_resources_c3_trajectory_jerk
      ((SFc3_trajectory_jerkInstanceStruct*) chartInstanceVar);
    ((SFc3_trajectory_jerkInstanceStruct*) chartInstanceVar)->
      ~SFc3_trajectory_jerkInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c3_trajectory_jerk(void *chartInstanceVar)
{
  mdl_start_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc3_trajectory_jerkInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c3_trajectory_jerk(void *chartInstanceVar)
{
  mdl_terminate_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c3_trajectory_jerk(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c3_trajectory_jerk((SFc3_trajectory_jerkInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c3_trajectory_jerk_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrdWFtv40QUdtJSKCrLchHsAxL7ABJCQixbod0HLu3mApEaGuG0SLxUE/s4ns14xjuXXBAv/BP",
    "+BX+AH8Ev4Jk3eEKccdw0ckLjcUAtWHLdY+ebc87ncxt7tU7Xw+MOnu/d87w9vL6AZ92bH8/lcm",
    "3pnN/f9T7J5d6LnhfEROoOj4TnfgQihCFw30QRnTpiuUl6RJJEVdDLSQJfgxLMaCq4m/GURyCBB",
    "7hAKqR20qtoYhjlo7bhgdWsvolpEPuxMCx8gguS8JSz2d/pTY3uocYmlRDoNkCoYynMMG4zMrye",
    "BaknjRiCkTKJM1cKtG9S66rqGqZpyqA1haDDlSbIgtrgr6+JhoaeOkcIVf4lWiQpo4SX5zomyoc",
    "Uo0PDWRri31Ojkb2SelHfgHKihaSEtRLWsBFeEttjaGcXw5o586wT3YSBGQ4pH1p2pUmAo/8YJy",
    "W4ihpiDJIM4ZQ75qD1rjXNXvAiLsvnYMeGZaUcNMn8rahK2Exva4wMqYp62wFvEMaUG7Yv0hMYA",
    "8v0N4kmFbBz/Q5gpWjYF+dE2ixwzCTD6TMDObYheEjLv+FxAZUV3K+weJaA08SGFIRI88L0xUKb",
    "YtIoLZIGplHz5KSkvlVsh2uQEQmgdO2ThCpAg7O4ctQbUkUGzKKRJZ15WXoF4JWhnooMb06EHCH",
    "HrkX2iiubCW5oCIfQBA1ZwWhhdJ8TZkranCiscDY8zhRWLDe9iLX5UwkckCCG0PYTyqALyi6gSt",
    "dn7CPH6O2Y6lkTVCBpWjaTjIIQG4llqT9L4YyPuJjwthSJn08E18QVAFYNIjm2hSfYluSsjcaXs",
    "1rCs76NrCrjWUI0IwMbG18Ax85ifbWdkASYVS2OoxsatA3Wp99ha+eKKo3j1KyV5UCYzaP3am7z",
    "6Cu5fLjIpY7qS3xTBNswx0npeJANe9CnCWQ3fIIzxVzMD6v3gXeld79+vd46/leriPO2xH25hNt",
    "dw8+rS7i7uRwcXmhJnmK+Cjm7eApytN7vgxJ8l8F5K7grfYt9xxK+tkavt3TdxNfOCl87Xq1Wy3",
    "BHS7iXCnp2C7i9nLPX/vzph58PXv7w0fu///r9p+++s018/Fh3i+c7ufzW5Xy26GDjlSJfJh7eK",
    "MSDlVVXn6jHIuqO6KPzNI76UefYn87X+21D/tUL9l7ev4+nxgKX1UsZdMJ8r2VlYuZ7ALv+4yV7",
    "9zbwsb8UT2jZ59vhXz8q4tfx9XyBLyunMb0Ise4ux+/N+XH/qBjX6/zYL/hhZR2DJpknq/l42+v",
    "fTdXbm8SVqcc7FXH1f7H+/5O4bf1z7TP/9d8/uKYueIXf373FflxXn1zmjdvm1y+e2zzwdi5/tv",
    "gW0IgpC9fsRvLHuGGI1j39n8T3H478vZnLLctf/jHz28NjTtgMtx/z7V1+uyftd7TFIwlErd/j3",
    "URfuLxu6vsHhfy28oTyUEzUBx89/PjhNv33L38+B68=",
    ""
  };

  static char newstr [1469] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c3_trajectory_jerk(SimStruct *S)
{
  const char* newstr = sf_c3_trajectory_jerk_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2176318995U));
  ssSetChecksum1(S,(789515460U));
  ssSetChecksum2(S,(3874957879U));
  ssSetChecksum3(S,(606420348U));
}

static void mdlRTW_c3_trajectory_jerk(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c3_trajectory_jerk(SimStruct *S)
{
  SFc3_trajectory_jerkInstanceStruct *chartInstance;
  chartInstance = (SFc3_trajectory_jerkInstanceStruct *)utMalloc(sizeof
    (SFc3_trajectory_jerkInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc3_trajectory_jerkInstanceStruct));
  chartInstance = new (chartInstance) SFc3_trajectory_jerkInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c3_trajectory_jerk;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c3_trajectory_jerk;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c3_trajectory_jerk;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c3_trajectory_jerk;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c3_trajectory_jerk;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c3_trajectory_jerk;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c3_trajectory_jerk;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c3_trajectory_jerk;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c3_trajectory_jerk;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c3_trajectory_jerk;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c3_trajectory_jerk;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c3_trajectory_jerk;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c3_trajectory_jerk(chartInstance);
}

void c3_trajectory_jerk_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c3_trajectory_jerk(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c3_trajectory_jerk(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c3_trajectory_jerk(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c3_trajectory_jerk_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
