/* Include files */

#include "trajectory_jerk_sfun.h"
#include "c14_trajectory_jerk.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include "stdio.h"
#include "string.h"
#include "stdlib.h"

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void c14_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, const real_T c14_b_attitude[3], real_T c14_b_R_BI[9]);
static void initialize_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance);
static void initialize_params_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance);
static void mdl_start_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance);
static void mdl_terminate_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance);
static void enable_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance);
static void disable_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance);
static void sf_gateway_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance);
static void ext_mode_exec_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance);
static void set_sim_state_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance, const mxArray *c14_st);
static void c14_emlrt_marshallIn(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, const mxArray *c14_nullptr, const char_T *c14_identifier,
  real_T c14_y[9]);
static void c14_b_emlrt_marshallIn(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId,
  real_T c14_y[9]);
static void c14_raiseCudaError(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, int32_T c14_errCode, const char_T *c14_file, uint32_T c14_line,
  const char_T *c14_errorName, const char_T *c14_errorString);
static void c14_checkCudaError(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, hipError_t c14_errCode, const char_T *c14_file, uint32_T
  c14_line);
static void init_dsm_address_info(SFc14_trajectory_jerkInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc14_trajectory_jerkInstanceStruct
  *chartInstance);

/* Function Definitions */
static void c14_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, const real_T c14_b_attitude[3], real_T c14_b_R_BI[9])
{
  c14_b_R_BI[0] = muDoubleScalarCos(c14_b_attitude[1]) * muDoubleScalarCos
    (c14_b_attitude[2]);
  c14_b_R_BI[3] = muDoubleScalarCos(c14_b_attitude[1]) * muDoubleScalarSin
    (c14_b_attitude[2]);
  c14_b_R_BI[6] = -muDoubleScalarSin(c14_b_attitude[1]);
  c14_b_R_BI[1] = -muDoubleScalarCos(c14_b_attitude[0]) * muDoubleScalarSin
    (c14_b_attitude[2]) + muDoubleScalarSin(c14_b_attitude[0]) *
    muDoubleScalarSin(c14_b_attitude[1]) * muDoubleScalarCos(c14_b_attitude[2]);
  c14_b_R_BI[4] = muDoubleScalarCos(c14_b_attitude[0]) * muDoubleScalarCos
    (c14_b_attitude[2]) + muDoubleScalarSin(c14_b_attitude[0]) *
    muDoubleScalarSin(c14_b_attitude[1]) * muDoubleScalarSin(c14_b_attitude[2]);
  c14_b_R_BI[7] = muDoubleScalarSin(c14_b_attitude[0]) * muDoubleScalarCos
    (c14_b_attitude[1]);
  c14_b_R_BI[2] = muDoubleScalarSin(c14_b_attitude[0]) * muDoubleScalarSin
    (c14_b_attitude[2]) + muDoubleScalarCos(c14_b_attitude[0]) *
    muDoubleScalarSin(c14_b_attitude[1]) * muDoubleScalarCos(c14_b_attitude[2]);
  c14_b_R_BI[5] = -muDoubleScalarSin(c14_b_attitude[0]) * muDoubleScalarCos
    (c14_b_attitude[2]) + muDoubleScalarCos(c14_b_attitude[0]) *
    muDoubleScalarSin(c14_b_attitude[1]) * muDoubleScalarSin(c14_b_attitude[2]);
  c14_b_R_BI[8] = muDoubleScalarCos(c14_b_attitude[0]) * muDoubleScalarCos
    (c14_b_attitude[1]);
}

static void initialize_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance)
{
  emlrtInitGPU(chartInstance->c14_fEmlrtCtx);
  hipGetLastError();
  emlrtLicenseCheckR2022a(chartInstance->c14_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void initialize_params_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance)
{
}

static void mdl_start_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance)
{
  hipError_t c14_errCode;
  c14_errCode = hipGetLastError();
  if (c14_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c14_errCode), (char_T *)
                       hipGetErrorName(c14_errCode), (char_T *)
                       hipGetErrorString(c14_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c14_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, &chartInstance->c14_JITStateAnimation
                        [0], &chartInstance->c14_JITTransitionAnimation[0]);
}

static void mdl_cleanup_runtime_resources_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance)
{
}

static void enable_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c14_trajectory_jerk(SFc14_trajectory_jerkInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c14_JITTransitionAnimation[0] = 0U;
  c14_c14_trajectory_jerk(chartInstance, *chartInstance->c14_attitude,
    *chartInstance->c14_R_BI);
}

static void ext_mode_exec_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance)
{
  const mxArray *c14_b_y = NULL;
  const mxArray *c14_st = NULL;
  const mxArray *c14_y = NULL;
  c14_st = NULL;
  c14_y = NULL;
  sf_mex_assign(&c14_y, sf_mex_createcellmatrix(1, 1), false);
  c14_b_y = NULL;
  sf_mex_assign(&c14_b_y, sf_mex_create("y", *chartInstance->c14_R_BI, 0, 0U, 1,
    0U, 2, 3, 3), false);
  sf_mex_setcell(c14_y, 0, c14_b_y);
  sf_mex_assign(&c14_st, c14_y, false);
  return c14_st;
}

static void set_sim_state_c14_trajectory_jerk
  (SFc14_trajectory_jerkInstanceStruct *chartInstance, const mxArray *c14_st)
{
  const mxArray *c14_u;
  real_T c14_dv[9];
  int32_T c14_i;
  c14_u = sf_mex_dup(c14_st);
  c14_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c14_u, 0)),
                       "R_BI", c14_dv);
  for (c14_i = 0; c14_i < 9; c14_i++) {
    (*chartInstance->c14_R_BI)[c14_i] = c14_dv[c14_i];
  }

  sf_mex_destroy(&c14_u);
  sf_mex_destroy(&c14_st);
}

const mxArray *sf_c14_trajectory_jerk_get_eml_resolved_functions_info()
{
  const mxArray *c14_nameCaptureInfo = NULL;
  c14_nameCaptureInfo = NULL;
  sf_mex_assign(&c14_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1, 0U, 2, 0, 1), false);
  return c14_nameCaptureInfo;
}

static void c14_emlrt_marshallIn(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, const mxArray *c14_nullptr, const char_T *c14_identifier,
  real_T c14_y[9])
{
  emlrtMsgIdentifier c14_thisId;
  c14_thisId.fIdentifier = const_cast<const char_T *>(c14_identifier);
  c14_thisId.fParent = NULL;
  c14_thisId.bParentIsCell = false;
  c14_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c14_nullptr), &c14_thisId,
    c14_y);
  sf_mex_destroy(&c14_nullptr);
}

static void c14_b_emlrt_marshallIn(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId,
  real_T c14_y[9])
{
  real_T c14_dv[9];
  int32_T c14_i;
  sf_mex_import(c14_parentId, sf_mex_dup(c14_u), c14_dv, 1, 0, 0U, 1, 0U, 2, 3,
                3);
  for (c14_i = 0; c14_i < 9; c14_i++) {
    c14_y[c14_i] = c14_dv[c14_i];
  }

  sf_mex_destroy(&c14_u);
}

static void c14_raiseCudaError(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, int32_T c14_errCode, const char_T *c14_file, uint32_T c14_line,
  const char_T *c14_errorName, const char_T *c14_errorString)
{
  emlrtRTEInfo c14_rtInfo;
  uint64_T c14_len;
  char_T *c14_brk;
  char_T *c14_fn;
  char_T *c14_pn;
  c14_len = strlen(c14_file);
  c14_pn = static_cast<char_T *>(calloc(c14_len + 1ULL, 1ULL));
  c14_fn = static_cast<char_T *>(calloc(c14_len + 1ULL, 1ULL));
  memcpy(c14_pn, c14_file, c14_len);
  memcpy(c14_fn, c14_file, c14_len);
  c14_brk = strrchr(c14_fn, '.');
  *c14_brk = '\x00';
  c14_brk = NULL;
  c14_brk = strrchr(c14_fn, '/');
  if (c14_brk == NULL) {
    c14_brk = strrchr(c14_fn, '\\');
  }

  if (c14_brk == NULL) {
    c14_brk = c14_fn;
  } else {
    c14_brk++;
  }

  c14_rtInfo.lineNo = c14_line;
  c14_rtInfo.colNo = 0;
  c14_rtInfo.fName = c14_brk;
  c14_rtInfo.pName = c14_pn;
  emlrtCUDAError(static_cast<uint32_T>(c14_errCode), (char_T *)c14_errorName,
                 (char_T *)c14_errorString, &c14_rtInfo,
                 chartInstance->c14_fEmlrtCtx);
}

static void c14_checkCudaError(SFc14_trajectory_jerkInstanceStruct
  *chartInstance, hipError_t c14_errCode, const char_T *c14_file, uint32_T
  c14_line)
{
  if (c14_errCode != hipSuccess) {
    c14_raiseCudaError(chartInstance, c14_errCode, c14_file, c14_line,
                       hipGetErrorName(c14_errCode), hipGetErrorString
                       (c14_errCode));
  }
}

static void init_dsm_address_info(SFc14_trajectory_jerkInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc14_trajectory_jerkInstanceStruct
  *chartInstance)
{
  chartInstance->c14_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c14_R_BI = (real_T (*)[9])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c14_attitude = (real_T (*)[3])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c14_trajectory_jerk_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(17653799U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2306842680U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2894203725U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3773306147U);
}

mxArray *sf_c14_trajectory_jerk_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c14_trajectory_jerk_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c14_trajectory_jerk_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c14_trajectory_jerk(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCWYGRgYAPSHEDMxAABrFA+IxKGiLPAxRWAuKSyIBUkXlyU7JkCpPMSc8H8xNI"
    "Kz7y0fLD5FgwI89kImM8JFYeAD/aU6RdxAOk3QNLPgkU/C5J+ASAvKN7Jkw/KBwA0MA19"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c14_trajectory_jerk_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sCFyIbuCo72dCr8xF114prE";
}

static void sf_opaque_initialize_c14_trajectory_jerk(void *chartInstanceVar)
{
  initialize_params_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
  initialize_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c14_trajectory_jerk(void *chartInstanceVar)
{
  enable_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c14_trajectory_jerk(void *chartInstanceVar)
{
  disable_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c14_trajectory_jerk(void *chartInstanceVar)
{
  sf_gateway_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c14_trajectory_jerk(SimStruct* S)
{
  return get_sim_state_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c14_trajectory_jerk(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c14_trajectory_jerk(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc14_trajectory_jerkInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_trajectory_jerk_optimization_info();
    }

    mdl_cleanup_runtime_resources_c14_trajectory_jerk
      ((SFc14_trajectory_jerkInstanceStruct*) chartInstanceVar);
    ((SFc14_trajectory_jerkInstanceStruct*) chartInstanceVar)->
      ~SFc14_trajectory_jerkInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c14_trajectory_jerk(void *chartInstanceVar)
{
  mdl_start_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc14_trajectory_jerkInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c14_trajectory_jerk(void *chartInstanceVar)
{
  mdl_terminate_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c14_trajectory_jerk(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c14_trajectory_jerk((SFc14_trajectory_jerkInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c14_trajectory_jerk_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [20] = {
    "eNrdWM2P20QUn6zCiqKyKgKpPVSCAwcuSOy2iCIh2l0ngUi7bFRni8RlNbFf4mnGM+585IMjJ/4",
    "sxKl/AAf+Bk4cucEbx8mGrNn1OFK3xZLjPDu/ee/95n05pNE9IXjs4dm7S8guXt/Gc4csjrcKub",
    "F2Lu43yVeF3HuHkCihynTFUBL/I5IxjECEdjhkM0+ssGmPKprqGnoFTeEpaMmtYVL4Gc/EEBSIC",
    "BfIpDJeejVLLWdi3LEicpr19wmLkjCRlsdHuCCNTwWf/5fezJoeamwxBZHpAMQmUdKOkg6no6tZ",
    "UGYaJBCNtU29udJgQps5V/WJ5YZlHNoziLpCG4os6Gv8DQ01EJiZd4QwHS7RMs04o6I61wnVIWQ",
    "YHQbOshg/T61B9irqRX0DJqiRilHeTnngIrwitsfRzhMMa+7Ns0lNCwZ2NGJi5NhVNgWB/mOcVO",
    "BqGMgJKDqCU+GZg8679izf4FVcVs/BrgvLWjlo08Wu6FrYXG97ggzpmno7kQgo59oP25fZMUyA5",
    "/pb1NAa2IV+D7DWLO7LZ1S5LPDMJCvYCwsFNpAiZtV3eLKBygvud1g8K8BZ6kIKYqR5Zfpqoeti",
    "0moj0wDTqHV8XFHfZWxXGFBDGkHl2qco04AG53HlqTdmmg64QyNLJvey8gogakOJHlrRmko1Ro5",
    "9i+wFVy4T/NAQj6AFBvKC0cbofka5rWhzqrHCufA401ix/PQi1uVPLXBEowRi108YhxPQbgFduT",
    "5jHzlEbyfMzFugI8WyqplkNcTYSBxL/XkGZ2Is5FR0lEzDYiK4Iq4AsGpQJbAtHGFbUvMOGl/Na",
    "gUv+i6y6oxnKTWcDlxsfAMCO4vz1XVCGmFWtQWObmjQNtiQ/YitXWimDY5T83aeA3E+j95r+M2j",
    "7xXyg1UudXVf4U5RbMMCJ6XDQT7sQZ+lkN8IKc4UC7E4nN7PyIXeWztX693Bb42aOLIl7ts1XLO",
    "En/fXcHcKOdp/eG4UfY4JK9X8/DmocakdtyvwXQVHLuEueF5eP1nDN0r0krXrNvvzZA337oae5g",
    "Zut+Ds55+Gf5uP7/+Rfvnyt19+ncTb6H/pGc97hXx/OZ+tOtjkUpGvEg93N+LByTrozLsDG8gvD",
    "uJAPZp19vcfZqpdK/+W9z/C02CBy+ulirpx8a7lZGoX7wBu/Udr9u5es/6ttXgi5M/H2+E/eLK5",
    "j2V8Nf/FV5M8PT/q7r1BdeKm6tJN4qrUrcYrxpFXjNvWP996/Kb//qo6QDZ+f+c19mPzWrcvv25",
    "+/U78+tCHhfz16p05SBiPS6b24jEO1sOyp/+T+P7Lk7/lnNB2/BV/+v3w4FBQPscxffEaVNzuKf",
    "d/0+qRAqrL34Vuoi+QkrmvrM/f3shvJ0+ZiOVUf7p/8PnBNv33Hzy/AGA=",
    ""
  };

  static char newstr [1409] = "";
  newstr[0] = '\0';
  for (i = 0; i < 20; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c14_trajectory_jerk(SimStruct *S)
{
  const char* newstr = sf_c14_trajectory_jerk_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(4284908168U));
  ssSetChecksum1(S,(3927712884U));
  ssSetChecksum2(S,(3569891693U));
  ssSetChecksum3(S,(1685504704U));
}

static void mdlRTW_c14_trajectory_jerk(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c14_trajectory_jerk(SimStruct *S)
{
  SFc14_trajectory_jerkInstanceStruct *chartInstance;
  chartInstance = (SFc14_trajectory_jerkInstanceStruct *)utMalloc(sizeof
    (SFc14_trajectory_jerkInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc14_trajectory_jerkInstanceStruct));
  chartInstance = new (chartInstance) SFc14_trajectory_jerkInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c14_trajectory_jerk;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c14_trajectory_jerk;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c14_trajectory_jerk;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c14_trajectory_jerk;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c14_trajectory_jerk;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c14_trajectory_jerk;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c14_trajectory_jerk;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c14_trajectory_jerk;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c14_trajectory_jerk;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c14_trajectory_jerk;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c14_trajectory_jerk;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c14_trajectory_jerk;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c14_trajectory_jerk(chartInstance);
}

void c14_trajectory_jerk_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c14_trajectory_jerk(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c14_trajectory_jerk(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c14_trajectory_jerk(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c14_trajectory_jerk_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
